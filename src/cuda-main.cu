#include <iostream>
#include <hipblas.h>
#include <vector>

#include "cuda_helper.h"
#include "lanch_kernel.cuh"

using namespace std;

int main(int argc, char **argv)
{
  if (argc != 2)
  {
    cout << "Usage: ./sgemm <kernel_id>" << endl;
    exit(1);
  }

  vector<int> test_cases = {1024, 2048, 4096};
  int max_size = test_cases[test_cases.size() - 1];
  int element_size = sizeof(float);
  int kernel_id = atoi(argv[1]);

  hipblasHandle_t err;
  hipblasCreate(&err);
  float alpha = 1.0f;
  float beta = 0.0f;

  float *a = (float *)malloc(max_size * max_size * element_size);
  float *b = (float *)malloc(max_size * max_size * element_size);
  float *c = (float *)malloc(max_size * max_size * element_size);
  float *c_ref = (float *)malloc(max_size * max_size * element_size);

  memset(a, 0, max_size * max_size * element_size);
  memset(b, 0, max_size * max_size * element_size);
  memset(c, 0, max_size * max_size * element_size);
  memset(c_ref, 0, max_size * max_size * element_size);

  float *dev_a, *dev_b, *dev_c, *dev_c_ref;
  CHECK(hipMalloc((void **)&dev_a, max_size * max_size * element_size));
  CHECK(hipMalloc((void **)&dev_b, max_size * max_size * element_size));
  CHECK(hipMalloc((void **)&dev_c, max_size * max_size * element_size));
  CHECK(hipMalloc((void **)&dev_c_ref, max_size * max_size * element_size));

  initialData(a, max_size * max_size);
  initialData(b, max_size * max_size);

  CHECK(hipMemcpy(dev_a, a, max_size * max_size, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(dev_b, b, max_size * max_size, hipMemcpyHostToDevice));
  CHECK(hipMemset(dev_c, 0, max_size * max_size * element_size));
  CHECK(hipMemset(dev_c_ref, 0, max_size * max_size * element_size));
  hipDeviceSynchronize();

  if (kernel_id != 0)
  {
    for (auto size : test_cases)
    {
      hipblasSgemm(err, HIPBLAS_OP_N, HIPBLAS_OP_N, size, size, size, &alpha, dev_a, size, dev_b, size, &beta, dev_c_ref, size);
      hipDeviceSynchronize();
      CHECK(hipMemcpy(c_ref, dev_c_ref, size * size, hipMemcpyDeviceToHost));
      hipDeviceSynchronize();
      lanch_kernel(kernel_id, dev_a, dev_b, dev_c, size, size, size);
      hipDeviceSynchronize();
      CHECK(hipMemcpy(c, dev_c, size * size, hipMemcpyDeviceToHost));
      hipDeviceSynchronize();
      checkResult(c_ref, c, size * size);
    }
  }
  else
  {
    for (auto size : test_cases)
    {
      lanch_kernel(0, dev_a, dev_b, dev_c, size, size, size);
      hipDeviceSynchronize();
    }
  }

  CHECK(hipFree(dev_a));
  CHECK(hipFree(dev_b));
  CHECK(hipFree(dev_c));
  CHECK(hipFree(dev_c_ref));

  free(a);
  free(b);
  free(c);
  free(c_ref);

  return 0;
}
