#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "lanch_kernel.cuh"

#define CEIL_DIV(x, y) (((x) + (y) - 1) / (y))

void call_my_kernel(int kernel_id, float *dev_a, float *dev_b, float *dev_c, int m, int k, int n) {
  dim3 block(32, 32);
  dim3 grid(CEIL_DIV(m, block.x), CEIL_DIV(n, block.y));
  switch (kernel_id)
  {
  case 1:
    mysgemm_v1<<<grid, block>>>(m, k, n, dev_a, dev_b, dev_c);
    break;
  
  default:
    break;
  }
}

void lanch_kernel(int kernel_id, float *dev_a, float *dev_b, float *dev_c, int m, int k, int n) {

  float alpha = 1.0f;
  float beta = 0.0f;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop); 

  hipEventRecord(start, 0);
  if (kernel_id == 0) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dev_a, m, dev_b, k, &beta, dev_c, m);
    hipblasDestroy(handle);
  } else {
    call_my_kernel(kernel_id, dev_a, dev_b, dev_c, m, k, n);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf( "Size:  %d * %d \t\tTime: %f ms \t\tPerformance: %f GFLOPS.\n", m, m, elapsedTime, 2.0*m*m*m/(elapsedTime*1e6));
}